#include "hip/hip_runtime.h"
__device__ void swap(int& a, int& b) {
    int t = a;
    a = b;
    b = t;
}
 
__global__ void sort(int* a, int flag_j, int flag_i, int count)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tid_comp = tid ^ flag_j;
 
    if (tid > count)
        return;
 
 
    if (tid_comp > tid) {
        if ((tid & flag_i) == 0) { //ascending
            if (a[tid] > a[tid_comp]) {
                swap(a[tid], a[tid_comp]);
            }
        }
        else { //desending
            if (a[tid] < a[tid_comp]) {
                swap(a[tid], a[tid_comp]);
            }
        }
    }
 
}
 
int main() 
{
 
 
    int count = 800;
    std::default_random_engine e;
 
    int* a, * b;
    hipHostMalloc((void**)&a, sizeof(int) * count);
    hipHostMalloc((void**)&b, sizeof(int) * count);
 
    for (int i = 0; i < count; i++)
    {
        a[i] = e() % 100000;
        //printf("a[%d], %d\n", i, a[i]);
    }
    int* d_a, * d_b;
    hipMallocManaged((void**)&d_a, sizeof(int) * count);
    hipMallocManaged((void**)&d_b, sizeof(int) * count);
    hipMemcpy(d_a, a, sizeof(int) * count, hipMemcpyHostToDevice);
 
 
    int thread = 512;
    int block = (count + thread) / thread;
 
    int _len = 1;
    while (_len < count)
    {
        _len <<= 1;
        printf("%d \n ", _len);
    }
 
 
 
    for (unsigned int i = 2; i <= _len; i <<= 1) {
        for (unsigned int j = i >> 1; j > 0; j >>= 1) {
 
            sort << <block, thread >> > (d_a, j, i, count);
            hipDeviceSynchronize();
        }
    }
 
    hipMemcpy(a, d_a, sizeof(int) * count, hipMemcpyDeviceToHost);
 
 
    printf("sorted\n");
    for (int i = 0; i < count; i++)
    {
        printf("a[%d], %d\n", i, a[i]);
    }

}